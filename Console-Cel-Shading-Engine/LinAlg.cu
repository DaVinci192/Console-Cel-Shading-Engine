#include "hip/hip_runtime.h"
﻿#include <iostream>

#include <hip/hip_runtime.h>
#include <cassert>

using namespace std;
#include <vector>;
#include <algorithm>;

#include ""

#include <wchar.h>
#include <Windows.h>

/*
void setupTerminal(int width, int height, HANDLE& hCon)
{
	hCon = CreateConsoleScreenBuffer(
		GENERIC_READ | GENERIC_WRITE,
		0,
		NULL,
		CONSOLE_TEXTMODE_BUFFER,
		NULL
	);
	SetConsoleActiveScreenBuffer(hCon);
	SetConsoleScreenBufferSize(hCon, { width, height });
}

void drawPixel(int x, int y, int width, int height, HANDLE hCon)
{
	COORD pos = { x, y };
	wchar_t* unicode_array = new wchar_t[width * height];
	unicode_array[x + y * width] = 0x2584;
	wchar_t unicode_text[1] = { 0x2584 };
	LPCWSTR str = unicode_text;
	DWORD len = 1;
	DWORD dwBytesWritten = 0;
	WriteConsoleOutputCharacter(hCon, str, len, pos, &dwBytesWritten);
}
*/

// CUDA kernel for vector addition
// __global__ means this is called from the CPU and is run on the GPU
__global__ void vectorAdd(int* a, int* b, int* c, int N)
{
	// calculate global thread ID
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	// boundary check
	if (tid < N)
	{
		// Each thread adds a single element
		c[tid] = a[tid] + b[tid];
	}
}

// copied from https://github.com/CoffeeBeforeArch/cuda_programming/blob/master/vectorAdd/baseline/vectorAdd.cu
void verify_result(std::vector<int>& a, std::vector<int>& b,
	std::vector<int>& c) {
	for (int i = 0; i < a.size(); i++) {
		assert(c[i] == a[i] + b[i]);
	}
}

int test()
{
	// array size 2^16
	constexpr int N = 1 << 16;
	size_t bytes = sizeof(int) * N;

	// vectors holding host-side (CPU-bound) data
	vector<int> a(N);
	vector<int> b(N);
	vector<int> c(N);

	// initialize random numbers in each array
	std::generate(begin(a), end(a), []() {return rand() % 100; });
	std::generate(begin(b), end(b), []() {return rand() % 100; });

	// allocate memory on the device 
	int* d_a, * d_b, * d_c;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// copy data from host device (CPU -> GPU)
	hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

	// Threads per CTA (1024 threads per CTA)
	int NUM_THREADS = 1 << 10;

	// CTAs per Grid
	// We need to launch at least as many threads as we have elements
	// this equation pads an extra CTA to the grid if N cannot be evenly divided
	// by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
	int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

	// Launch the kernel on the GPU
	// Kernel calls are asnyc (the CPU program continues execution after the call, no
	// necessarily before the kernel finishes
	vectorAdd <<<NUM_BLOCKS, NUM_THREADS>>> (d_a, d_b, d_c, N);

	// copy sum vector from device to host 
	// hipMemcpy is synchronous, waits for the prior kernel
	// launch to complete (both go to the default stream in this case).
	// Therefore hipMemcpy acts as both a memcpy and synchronization barrier
	hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

	// check results for errors
	verify_result(a, b, c);

	// free memory on device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	cout << "Completed Successfully" << endl;

	return 0;

}
