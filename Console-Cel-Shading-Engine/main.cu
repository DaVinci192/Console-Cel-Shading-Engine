﻿#include <iostream>

#include <hip/hip_runtime.h>
#include <cassert>

#include <vector>;
#include <algorithm>;

#include ""

#include <wchar.h>
#include <Windows.h>

#include "algebra.cuh"
#include "terminal.cuh"

#include <stdlib.h>
#include <time.h>

using namespace algebra;

// cmake generates a solution file
// cmake findPackage

/*
namespace dims
{
	const SHORT WIDTH = 140;
	const SHORT HEIGHT = 70;
	const SHORT SIZE = WIDTH * HEIGHT;
}
*/

void drawNoise(Terminal screen)
{
	for (int i = 0; i < dims::HEIGHT; i++)
	{
		for (int j = 0; j < dims::WIDTH; j++)
		{
			screen.drawPixel(i, j, rand() % 256);
		}
	}
}

int main()
{
	srand(time(0));

	CHAR_INFO ptr[dims::SIZE];

	
	Terminal Screen = Terminal(ptr, 15);
	

	if (!Screen.activate())
	{
		std::cout << "there was an error" << std::endl;
	}
	Screen.clear();
	
	
	
	while (1)
	{
		drawNoise(Screen);
		Screen.draw();
		Screen.clear();
	}

	//wprintf(L"\x1b[31mThis text has a red foreground using SGR.31.\r\n");
	//wprintf(L"\x1b[38;2;255;0;0;48;2;0;255;0m▄");
	//printf("\u2580");

	//std::cout << "\u2580" << std::endl;

}
